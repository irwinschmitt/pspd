#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void sumKernel(int *d_arr, int *d_sum, int n) {
  extern __shared__ int sdata[];
  unsigned int tid = threadIdx.x;
  unsigned int i = blockIdx.x * blockDim.x + tid;

  sdata[tid] = (i < n) ? d_arr[i] : 0;
  __syncthreads();

  for (unsigned int s = blockDim.x / 2; s > 0; s >>= 1) {
    if (tid < s) {
      sdata[tid] += sdata[tid + s];
    }
    __syncthreads();
  }

  if (tid == 0) {
    atomicAdd(d_sum, sdata[0]);
  }
}

int main() {
  int n;
  scanf("%d", &n);

  int size = 1 << n;  // 2^n
  int *h_arr = (int *)malloc(size * sizeof(int));
  int h_sum = 0;

  for (int i = 0; i < size; i++) {
    h_arr[i] = i;
  }

  int *d_arr, *d_sum;
  hipMalloc((void **)&d_arr, size * sizeof(int));
  hipMalloc((void **)&d_sum, sizeof(int));
  hipMemcpy(d_arr, h_arr, size * sizeof(int), hipMemcpyHostToDevice);

  int threadsPerBlock = 1024;
  int blocksPerGrid = (size + threadsPerBlock - 1) / threadsPerBlock;

  sumKernel<<<blocksPerGrid, threadsPerBlock, threadsPerBlock * sizeof(int)>>>(
      d_arr, d_sum, size);

  hipDeviceSynchronize();

  hipMemcpy(&h_sum, d_sum, sizeof(int), hipMemcpyDeviceToHost);

  printf("%d\n", h_sum);

  hipFree(d_arr);
  hipFree(d_sum);
  free(h_arr);

  return 0;
}
